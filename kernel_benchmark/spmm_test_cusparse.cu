/***************************************************************************
 * Copyright 2023 The FLash-LLM Authors. All rights reserved.
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * http://www.apache.org/licenses/LICENSE-2.0
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 ***************************************************************************/


#define USE_CUSPARSE

#include "./spmm_test_utils.h"
#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
int main(int argc, char** argv)
{
    if (argc != 6) {
        printf("Wrong Inputs! Correct input format: ./spmm_test M K N Sparsity SplitK\n");
        return;
    }
    int M_GLOBAL                    = atoi(argv[1]);
    int K_GLOBAL                    = atoi(argv[2]);
    int N_GLOBAL                    = atoi(argv[3]);
    int MATRIX_A_PRUNING_PERCENTAGE = atoi(argv[4]);
    int SPLIT_K                     = atoi(argv[5]);
    //
    // printf("M: %d N: %d K: %d\n", M_GLOBAL, N_GLOBAL, K_GLOBAL);
    //
    hipblasStatus_t cublas_status;
    // hipsparseStatus_t  cusparse_status;
    // hipError_t       cuda_error;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Host memory
    half* A_h            = NULL;  // row major
    half* B_h            = NULL;  // col major
    half* B_Transposed_h = NULL;  // row major
    // Device memory
    half* A            = NULL;
    half* B            = NULL;
    half* B_Transposed = NULL;
    //
    A_h            = (half*)malloc(sizeof(half) * M_GLOBAL * K_GLOBAL);
    B_h            = (half*)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL);
    B_Transposed_h = (half*)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL);
    if (A_h == NULL || B_h == NULL || B_Transposed_h == NULL) {
        printf("Error in CPU Malloc!\n");
        exit(-1);
    }
    hipMalloc(reinterpret_cast<void**>(&A), sizeof(half) * M_GLOBAL * K_GLOBAL);
    hipMalloc(reinterpret_cast<void**>(&B), sizeof(half) * N_GLOBAL * K_GLOBAL);
    hipMalloc(reinterpret_cast<void**>(&B_Transposed), sizeof(half) * N_GLOBAL * K_GLOBAL);
    checkLastCudaError(__LINE__);
    if (A == NULL || B == NULL || B_Transposed == NULL) {
        printf("Error in hipMalloc!\n");
        exit(-1);
    }
    //
    init_host_matrices(A_h, B_h, M_GLOBAL, K_GLOBAL, N_GLOBAL, MATRIX_A_PRUNING_PERCENTAGE);
    for (int i = 0; i < K_GLOBAL; i++)
        for (int j = 0; j < N_GLOBAL; j++)
            B_Transposed_h[i * N_GLOBAL + j] = B_h[i + j * K_GLOBAL];
    //
    // printf("Preparing dense data for GPU...\n");
    hipMemcpy(A, A_h, sizeof(half) * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    hipMemcpy(B, B_h, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    hipMemcpy(B_Transposed, B_Transposed_h, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    checkLastCudaError(__LINE__);
  
#ifdef USE_CUSPARSE
    /////////////////////////////////////////////////////////////////////////////////////////////////
    printf("Launching CuSparse_ColMajor...\n");
    half* D_CuSparse = NULL;
    hipMalloc(&D_CuSparse, N_GLOBAL * M_GLOBAL * sizeof(half));
    if (D_CuSparse == NULL) {
        printf("Error in spmm_test.cu: line %d hipMalloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemset(D_CuSparse, 0.0f, N_GLOBAL * M_GLOBAL * sizeof(half));
    //
    hipsparseHandle_t sp_handle = 0;
    hipsparseCreate(&sp_handle);
    hipsparseSetStream(sp_handle, 0);
    hipsparseSpMatDescr_t SpMatA;
    hipsparseDnMatDescr_t DnMatA, DnMatB, DnMatC;
    // Create Dense Matrix
    CHECK_CUSPARSE(hipsparseCreateDnMat(&DnMatA,
                                       M_GLOBAL,
                                       K_GLOBAL,
                                       K_GLOBAL,
                                       A,
                                       HIP_R_16F,
                                       HIPSPARSE_ORDER_ROW))  // Very critical!!! Weight Matrix must be row major,
                                                             // otherwise causing significant performance problems
    CHECK_CUSPARSE(hipsparseCreateDnMat(&DnMatB, K_GLOBAL, N_GLOBAL, K_GLOBAL, B, HIP_R_16F, HIPSPARSE_ORDER_COL))
    CHECK_CUSPARSE(
        hipsparseCreateDnMat(&DnMatC, M_GLOBAL, N_GLOBAL, M_GLOBAL, D_CuSparse, HIP_R_16F, HIPSPARSE_ORDER_COL))
    // Create Sparse Matrix in CSR format
    int* csrRowPtr;
    hipMalloc(&csrRowPtr, sizeof(int) * (M_GLOBAL + 1));
    CHECK_CUSPARSE(hipsparseCreateCsr(&SpMatA,
                                     M_GLOBAL,
                                     K_GLOBAL,
                                     0,
                                     csrRowPtr,
                                     NULL,
                                     NULL,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO,
                                     HIP_R_16F))
    // execute Sparse to Dense conversion
    void*  Buffer     = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(
        hipsparseDenseToSparse_bufferSize(sp_handle, DnMatA, SpMatA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize))
    hipMalloc(&Buffer, bufferSize);
    CHECK_CUSPARSE(
        hipsparseDenseToSparse_analysis(sp_handle, DnMatA, SpMatA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, Buffer))
    //
    int64_t numRowTMP, numColTMP, NNZ_1;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(SpMatA, &numRowTMP, &numColTMP, &NNZ_1))
    //
    int*  csrColInd;
    half* csrVal;
    hipMalloc(&csrColInd, NNZ_1 * sizeof(int));
    hipMalloc(&csrVal, NNZ_1 * sizeof(half));
    //
    CHECK_CUSPARSE(hipsparseCsrSetPointers(SpMatA, csrRowPtr, csrColInd, csrVal))
    CHECK_CUSPARSE(hipsparseDenseToSparse_convert(sp_handle, DnMatA, SpMatA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, Buffer))
    //
    hipsparseSpMMAlg_t CuSparse_Algorithm;
    CuSparse_Algorithm = HIPSPARSE_SPMM_ALG_DEFAULT;
    CuSparse_Algorithm =
        HIPSPARSE_SPMM_CSR_ALG1;  // csrmm_kernel faster: Provide the best performance with column-major layout
    // CuSparse_Algorithm = HIPSPARSE_SPMM_CSR_ALG2;    // csrmm_v2_kernel: Provide the best performance with row-major
    // layout!!! How about try row major of B&C? CuSparse_Algorithm = HIPSPARSE_SPMM_CSR_ALG3;
    //
    // printf("CuSparse Algorithm: %d \n", CuSparse_Algorithm);
    //
    const float alpha_float = 1.0;
    const float beta_float  = 0.0;
    //
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(sp_handle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha_float,
                                           SpMatA,
                                           DnMatB,
                                           &beta_float,
                                           DnMatC,
                                           HIP_R_32F,
                                           CuSparse_Algorithm,
                                           &bufferSize))
    hipFree(Buffer);
    hipMalloc(&Buffer, bufferSize);
    for (int i = 0; i < CUSPARSE_ITERATION; i++)
        CHECK_CUSPARSE(hipsparseSpMM(sp_handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha_float,
                                    SpMatA,
                                    DnMatB,
                                    &beta_float,
                                    DnMatC,
                                    HIP_R_32F,
                                    CuSparse_Algorithm,
                                    Buffer))
    hipEventRecord(start);
    for (int i = 0; i < CUSPARSE_ITERATION; i++)
        CHECK_CUSPARSE(hipsparseSpMM(sp_handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha_float,
                                    SpMatA,
                                    DnMatB,
                                    &beta_float,
                                    DnMatC,
                                    HIP_R_32F,
                                    CuSparse_Algorithm,
                                    Buffer))
    hipEventRecord(stop);
    //
    float milliseconds_CuSparse_ColMajor = 0.0f;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds_CuSparse_ColMajor, start, stop);
    milliseconds_CuSparse_ColMajor = milliseconds_CuSparse_ColMajor / CUSPARSE_ITERATION;
    float tflops_CuSparse_ColMajor = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2)
                                                         / (milliseconds_CuSparse_ColMajor / 1000.))
                                     / 1e12;
    //
    half* D_CuSparse_h;
    D_CuSparse_h = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_CuSparse_h == NULL) {
        printf("Error in spmm_test.cu: line %d CPU Malloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemcpy(D_CuSparse_h, D_CuSparse, N_GLOBAL * M_GLOBAL * sizeof(half), hipMemcpyDeviceToHost);
    hipFree(D_CuSparse);
    hipFree(csrRowPtr);
    hipFree(csrColInd);
    hipFree(csrVal);
    hipFree(Buffer);
    /////////////////////////////////////////////////////////////////////////////////////////////////
    printf("Launching CuSparse_RowMajor...\n");
    half* D_CuSparse_2;
    hipMalloc(&D_CuSparse_2, N_GLOBAL * M_GLOBAL * sizeof(half));
    if (D_CuSparse_2 == NULL) {
        printf("Error in Test_SpMM_v2.cu: line %d hipMalloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemset(D_CuSparse_2, 0.0f, N_GLOBAL * M_GLOBAL * sizeof(half));
    //
    hipsparseHandle_t sp_handle_2 = 0;
    hipsparseCreate(&sp_handle_2);
    hipsparseSpMatDescr_t SpMatA_2;
    hipsparseDnMatDescr_t DnMatA_2, DnMatB_2, DnMatC_2;
    // Create Dense Matrix
    CHECK_CUSPARSE(hipsparseCreateDnMat(&DnMatA_2,
                                       M_GLOBAL,
                                       K_GLOBAL,
                                       K_GLOBAL,
                                       A,
                                       HIP_R_16F,
                                       HIPSPARSE_ORDER_ROW))  // Very critical!!! Weight Matrix must be row major,
                                                             // otherwise causing significant performance problems

    CHECK_CUSPARSE(
        hipsparseCreateDnMat(&DnMatB_2, K_GLOBAL, N_GLOBAL, N_GLOBAL, B_Transposed, HIP_R_16F, HIPSPARSE_ORDER_ROW))
    CHECK_CUSPARSE(
        hipsparseCreateDnMat(&DnMatC_2, M_GLOBAL, N_GLOBAL, N_GLOBAL, D_CuSparse_2, HIP_R_16F, HIPSPARSE_ORDER_ROW))
    // Create Sparse Matrix in CSR format
    int* csrRowPtr_2;
    hipMalloc(&csrRowPtr_2, sizeof(int) * (M_GLOBAL + 1));
    CHECK_CUSPARSE(hipsparseCreateCsr(&SpMatA_2,
                                     M_GLOBAL,
                                     K_GLOBAL,
                                     0,
                                     csrRowPtr_2,
                                     NULL,
                                     NULL,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO,
                                     HIP_R_16F))
    // execute Sparse to Dense conversion
    void*  Buffer_2     = NULL;
    size_t bufferSize_2 = 0;
    CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(
        sp_handle_2, DnMatA_2, SpMatA_2, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize_2))
    hipMalloc(&Buffer_2, bufferSize_2);
    CHECK_CUSPARSE(
        hipsparseDenseToSparse_analysis(sp_handle_2, DnMatA_2, SpMatA_2, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, Buffer_2))
    //
    int64_t numRowTMP_2, numColTMP_2, NNZ_2;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(SpMatA_2, &numRowTMP_2, &numColTMP_2, &NNZ_2))
    //
    int*  csrColInd_2;
    half* csrVal_2;
    hipMalloc(&csrColInd_2, NNZ_2 * sizeof(int));
    hipMalloc(&csrVal_2, NNZ_2 * sizeof(half));
    //
    CHECK_CUSPARSE(hipsparseCsrSetPointers(SpMatA_2, csrRowPtr_2, csrColInd_2, csrVal_2))
    CHECK_CUSPARSE(
        hipsparseDenseToSparse_convert(sp_handle_2, DnMatA_2, SpMatA_2, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, Buffer_2))
    //
    hipsparseSpMMAlg_t CuSparse_Algorithm_2;
    CuSparse_Algorithm_2 = HIPSPARSE_SPMM_ALG_DEFAULT;
    CuSparse_Algorithm_2 =
        HIPSPARSE_SPMM_CSR_ALG1;  // csrmm_kernel faster: Provide the best performance with column-major layout
    CuSparse_Algorithm_2 = HIPSPARSE_SPMM_CSR_ALG2;  // csrmm_v2_kernel: Provide the best performance with row-major
                                                    // layout!!! How about try row major of B&C?
    // CuSparse_Algorithm_2 = HIPSPARSE_SPMM_CSR_ALG3;
    // printf("CuSparse Algorithm: %d \n", CuSparse_Algorithm_2);
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(sp_handle_2,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha_float,
                                           SpMatA_2,
                                           DnMatB_2,
                                           &beta_float,
                                           DnMatC_2,
                                           HIP_R_32F,
                                           CuSparse_Algorithm_2,
                                           &bufferSize_2))
    hipFree(Buffer_2);
    hipMalloc(&Buffer_2, bufferSize_2);
    //
    for (int i = 0; i < CUSPARSE_ITERATION; i++)
        CHECK_CUSPARSE(hipsparseSpMM(sp_handle_2,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha_float,
                                    SpMatA_2,
                                    DnMatB_2,
                                    &beta_float,
                                    DnMatC_2,
                                    HIP_R_32F,
                                    CuSparse_Algorithm_2,
                                    Buffer_2))
    hipEventRecord(start);
    for (int i = 0; i < CUSPARSE_ITERATION; i++)
        CHECK_CUSPARSE(hipsparseSpMM(sp_handle_2,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha_float,
                                    SpMatA_2,
                                    DnMatB_2,
                                    &beta_float,
                                    DnMatC_2,
                                    HIP_R_32F,
                                    CuSparse_Algorithm_2,
                                    Buffer_2))
    hipEventRecord(stop);
    //
    float milliseconds_CuSparse_RowMajor;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds_CuSparse_RowMajor, start, stop);
    milliseconds_CuSparse_RowMajor = milliseconds_CuSparse_RowMajor / CUSPARSE_ITERATION;
    float tflops_CuSparse_RowMajor = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2)
                                                         / (milliseconds_CuSparse_RowMajor / 1000.))
                                     / 1e12;
    // transpose result to col-major
    half* D_CuSparse_h_2_row_major;
    half* D_CuSparse_h_2;
    D_CuSparse_h_2           = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    D_CuSparse_h_2_row_major = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_CuSparse_h_2 == NULL || D_CuSparse_h_2_row_major == NULL) {
        printf("Error in spmm_test.cu: line %d CPU Malloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemcpy(D_CuSparse_h_2_row_major,
               D_CuSparse_2,
               N_GLOBAL * M_GLOBAL * sizeof(half),
               hipMemcpyDeviceToHost);  // row major
    for (int i = 0; i < N_GLOBAL; i++)
        for (int j = 0; j < M_GLOBAL; j++)
            D_CuSparse_h_2[i * M_GLOBAL + j] = D_CuSparse_h_2_row_major[i + j * N_GLOBAL];
    free(D_CuSparse_h_2_row_major);
    hipFree(D_CuSparse_2);
    hipFree(csrRowPtr_2);
    hipFree(csrColInd_2);
    hipFree(csrVal_2);
    hipFree(Buffer_2);
    /////////////////////////////////////////////////////////////////////////////////////////////////
#endif



    printf("******************************************Problem Size******************************************\n");
    printf("M: %d N: %d K: %d Pruning Rate: %d SplitK: %d\n",
           M_GLOBAL,
           N_GLOBAL,
           K_GLOBAL,
           MATRIX_A_PRUNING_PERCENTAGE,
           SPLIT_K);
// printf("******************************************Performance*******************************************\n");

#ifdef USE_CUSPARSE
    PrintPerformance("CuSparse_C", milliseconds_CuSparse_ColMajor, tflops_CuSparse_ColMajor, 0.0);
    PrintPerformance("CuSparse_R", milliseconds_CuSparse_RowMajor, tflops_CuSparse_RowMajor, 0.0);
#endif

    free(A_h);
    free(B_h);
    free(B_Transposed_h);
    hipFree(A);
    hipFree(B);
    hipFree(B_Transposed);
    return 0;
}
