/***************************************************************************
 * Copyright 2025 The SpInfer Authors. All rights reserved.
 * Copyright 2023 The FLash-LLM Authors. All rights reserved.
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * http://www.apache.org/licenses/LICENSE-2.0
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 ***************************************************************************/
#define USE_CUSPARSE
#include "./spmm_test_utils.h"
#include <assert.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
int main(int argc, char** argv)
{
    if (argc != 6) {
        printf("Wrong Inputs! Correct input format: ./spmm_test M K N Sparsity SplitK\n");
        return;
    }
    int M_GLOBAL                    = atoi(argv[1]);
    int K_GLOBAL                    = atoi(argv[2]);
    int N_GLOBAL                    = atoi(argv[3]);
    int MATRIX_A_PRUNING_PERCENTAGE = atoi(argv[4]);
    int SPLIT_K                     = atoi(argv[5]);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Host memory
    half* A_h            = NULL;  // row major
    half* B_h            = NULL;  // col major
    half* B_Transposed_h = NULL;  // row major
    // Device memory
    half* A            = NULL;
    half* B            = NULL;
    half* B_Transposed = NULL;
    //
    A_h            = (half*)malloc(sizeof(half) * M_GLOBAL * K_GLOBAL);
    B_h            = (half*)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL);
    B_Transposed_h = (half*)malloc(sizeof(half) * K_GLOBAL * N_GLOBAL);
    if (A_h == NULL || B_h == NULL || B_Transposed_h == NULL) {
        printf("Error in CPU Malloc!\n");
        exit(-1);
    }
    hipMalloc(reinterpret_cast<void**>(&A), sizeof(half) * M_GLOBAL * K_GLOBAL);
    hipMalloc(reinterpret_cast<void**>(&B), sizeof(half) * N_GLOBAL * K_GLOBAL);
    hipMalloc(reinterpret_cast<void**>(&B_Transposed), sizeof(half) * N_GLOBAL * K_GLOBAL);
    checkLastCudaError(__LINE__);
    if (A == NULL || B == NULL || B_Transposed == NULL) {
        printf("Error in hipMalloc!\n");
        exit(-1);
    }
    //
    init_host_matrices(A_h, B_h, M_GLOBAL, K_GLOBAL, N_GLOBAL, MATRIX_A_PRUNING_PERCENTAGE);
    for (int i = 0; i < K_GLOBAL; i++)
        for (int j = 0; j < N_GLOBAL; j++)
            B_Transposed_h[i * N_GLOBAL + j] = B_h[i + j * K_GLOBAL];
    //
    // printf("Preparing dense data for GPU...\n");
    hipMemcpy(A, A_h, sizeof(half) * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    hipMemcpy(B, B_h, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    hipMemcpy(B_Transposed, B_Transposed_h, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice);
    checkLastCudaError(__LINE__);
  
    /////////////////////////////////////////////////////////////////////////////////////////////////
    printf("Launching CuSparse_ColMajor...\n");
    half* D_CuSparse = NULL;
    hipMalloc(&D_CuSparse, N_GLOBAL * M_GLOBAL * sizeof(half));
    if (D_CuSparse == NULL) {
        printf("Error in spmm_test.cu: line %d hipMalloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemset(D_CuSparse, 0.0f, N_GLOBAL * M_GLOBAL * sizeof(half));
    //
    hipsparseHandle_t sp_handle = 0;
    hipsparseCreate(&sp_handle);
    hipsparseSetStream(sp_handle, 0);
    hipsparseSpMatDescr_t SpMatA;
    hipsparseDnMatDescr_t DnMatA, DnMatB, DnMatC;
    // Create Dense Matrix
    CHECK_CUSPARSE(hipsparseCreateDnMat(&DnMatA,
                                       M_GLOBAL,
                                       K_GLOBAL,
                                       K_GLOBAL,
                                       A,
                                       HIP_R_16F,
                                       HIPSPARSE_ORDER_ROW))  // Very critical!!! Weight Matrix must be row major,
                                                             // otherwise causing significant performance problems
    CHECK_CUSPARSE(hipsparseCreateDnMat(&DnMatB, K_GLOBAL, N_GLOBAL, K_GLOBAL, B, HIP_R_16F, HIPSPARSE_ORDER_COL))
    CHECK_CUSPARSE(
        hipsparseCreateDnMat(&DnMatC, M_GLOBAL, N_GLOBAL, M_GLOBAL, D_CuSparse, HIP_R_16F, HIPSPARSE_ORDER_COL))
    // Create Sparse Matrix in CSR format
    int* csrRowPtr;
    hipMalloc(&csrRowPtr, sizeof(int) * (M_GLOBAL + 1));
    CHECK_CUSPARSE(hipsparseCreateCsr(&SpMatA,
                                     M_GLOBAL,
                                     K_GLOBAL,
                                     0,
                                     csrRowPtr,
                                     NULL,
                                     NULL,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO,
                                     HIP_R_16F))
    // execute Sparse to Dense conversion
    void*  Buffer     = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(
        hipsparseDenseToSparse_bufferSize(sp_handle, DnMatA, SpMatA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize))
    hipMalloc(&Buffer, bufferSize);
    CHECK_CUSPARSE(
        hipsparseDenseToSparse_analysis(sp_handle, DnMatA, SpMatA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, Buffer))
    //
    int64_t numRowTMP, numColTMP, NNZ_1;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(SpMatA, &numRowTMP, &numColTMP, &NNZ_1))
    //
    int*  csrColInd;
    half* csrVal;
    hipMalloc(&csrColInd, NNZ_1 * sizeof(int));
    hipMalloc(&csrVal, NNZ_1 * sizeof(half));
    //
    CHECK_CUSPARSE(hipsparseCsrSetPointers(SpMatA, csrRowPtr, csrColInd, csrVal))
    CHECK_CUSPARSE(hipsparseDenseToSparse_convert(sp_handle, DnMatA, SpMatA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, Buffer))
    //
    hipsparseSpMMAlg_t CuSparse_Algorithm;
    CuSparse_Algorithm = HIPSPARSE_SPMM_ALG_DEFAULT;
    CuSparse_Algorithm =
        HIPSPARSE_SPMM_CSR_ALG1;  // csrmm_kernel faster: Provide the best performance with column-major layout
    const float alpha_float = 1.0;
    const float beta_float  = 0.0;
    //
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(sp_handle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha_float,
                                           SpMatA,
                                           DnMatB,
                                           &beta_float,
                                           DnMatC,
                                           HIP_R_32F,
                                           CuSparse_Algorithm,
                                           &bufferSize))
    hipFree(Buffer);
    hipMalloc(&Buffer, bufferSize);
    for (int i = 0; i < CUSPARSE_ITERATION; i++)
        CHECK_CUSPARSE(hipsparseSpMM(sp_handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha_float,
                                    SpMatA,
                                    DnMatB,
                                    &beta_float,
                                    DnMatC,
                                    HIP_R_32F,
                                    CuSparse_Algorithm,
                                    Buffer))
    hipEventRecord(start);
    for (int i = 0; i < CUSPARSE_ITERATION; i++)
        CHECK_CUSPARSE(hipsparseSpMM(sp_handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    &alpha_float,
                                    SpMatA,
                                    DnMatB,
                                    &beta_float,
                                    DnMatC,
                                    HIP_R_32F,
                                    CuSparse_Algorithm,
                                    Buffer))
    hipEventRecord(stop);
    //
    float milliseconds_CuSparse_ColMajor = 0.0f;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds_CuSparse_ColMajor, start, stop);
    milliseconds_CuSparse_ColMajor = milliseconds_CuSparse_ColMajor / CUSPARSE_ITERATION;
    float tflops_CuSparse_ColMajor = static_cast<double>((static_cast<double>(M_GLOBAL) * N_GLOBAL * K_GLOBAL * 2)
                                                         / (milliseconds_CuSparse_ColMajor / 1000.))
                                     / 1e12;
    //
    half* D_CuSparse_h;
    D_CuSparse_h = (half*)malloc(sizeof(half) * M_GLOBAL * N_GLOBAL);
    if (D_CuSparse_h == NULL) {
        printf("Error in spmm_test.cu: line %d CPU Malloc falied\n", __LINE__);
        exit(-1);
    }
    hipMemcpy(D_CuSparse_h, D_CuSparse, N_GLOBAL * M_GLOBAL * sizeof(half), hipMemcpyDeviceToHost);
    hipFree(D_CuSparse);
    hipFree(csrRowPtr);
    hipFree(csrColInd);
    hipFree(csrVal);
    hipFree(Buffer);
    /////////////////////////////////////////////////////////////////////////////////////////////////

    printf("******************************************Problem Size******************************************\n");
    printf("M: %d N: %d K: %d Pruning Rate: %d SplitK: %d\n",
           M_GLOBAL,
           N_GLOBAL,
           K_GLOBAL,
           MATRIX_A_PRUNING_PERCENTAGE,
           SPLIT_K);
// printf("******************************************Performance*******************************************\n");

    PrintPerformance("CuSparse_C", milliseconds_CuSparse_ColMajor, tflops_CuSparse_ColMajor, 0.0);

    SaveCuSparsePerformanceData("cusparse_performance_results.csv",
        M_GLOBAL, K_GLOBAL, N_GLOBAL, 
        SPLIT_K, MATRIX_A_PRUNING_PERCENTAGE,
        milliseconds_CuSparse_ColMajor, tflops_CuSparse_ColMajor);

    free(A_h);
    free(B_h);
    free(B_Transposed_h);
    hipFree(A);
    hipFree(B);
    hipFree(B_Transposed);
    return 0;
}
